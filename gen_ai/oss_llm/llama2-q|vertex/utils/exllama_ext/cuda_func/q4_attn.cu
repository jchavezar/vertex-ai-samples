#include "hip/hip_runtime.h"
#include "q4_mlp.cuh"
#include "q4_matmul.cuh"
#include "rope.cuh"
#include "rms_norm.cuh"
#include "half_matmul.cuh"
#include "../cuda_buffers.cuh"
#include "../util.cuh"
#include "../matrix.cuh"
#if defined(USE_ROCM)
#include "../hip_compat.cuh"
#endif

const int THREADS_X = 32;
const int THREADS_Y = 1;
const int THREADS_Z = 4;
const int BLOCKSIZE_X = 2; // 2*half == 1*uint32_t
const int BLOCKSIZE_Z = 4; // num_heads must be divisible by BLOCKSIZE_Z  TODO: Check that this is the case when Llama2-34b releases

__global__ void update_cache_kernel
(
    const half* __restrict__ key_states,
    const half* __restrict__ value_states,
    half* __restrict__ key_cache,
    half* __restrict__ value_cache,
    const int head_dim,
    const int num_kv_heads,
    const int q_len,
    const int max_seq_len,
    const int past_len
)
{
    //int state_shape[]  = {              num_kv_heads,                     q_len, head_dim };
    int state_stride[] = {                  head_dim,   head_dim * num_kv_heads,        1 };
    int state_pos[]    = {                         0,                         0,        0 };

    //int cache_shape[]  = {              num_kv_heads,               max_seq_len, head_dim };
    int cache_stride[] = {    max_seq_len * head_dim,                  head_dim,        1 };
    int cache_pos[]    = {                         0,                  past_len,        0 };

    int size[]         = {              num_kv_heads,                  q_len, head_dim };

    int x = (blockIdx.x * THREADS_X + threadIdx.x) * BLOCKSIZE_X; 
    int y = blockIdx.y * THREADS_Y + threadIdx.y;
    int z = (blockIdx.z * THREADS_Z + threadIdx.z) * BLOCKSIZE_Z;
    
    if (x >= size[2]) return;
    if (y >= size[1]) return;
    if (z >= size[0]) return;

    int state_offset = (z + state_pos[0]) * state_stride[0] + (y + state_pos[1]) * state_stride[1] + (x + state_pos[2]) * state_stride[2];
    int cache_offset = (z + cache_pos[0]) * cache_stride[0] + (y + cache_pos[1]) * cache_stride[1] + (x + cache_pos[2]) * cache_stride[2];

    const uint32_t* key_ptr = (uint32_t*) (key_states + state_offset);
    const uint32_t* value_ptr = (uint32_t*) (value_states + state_offset);
    uint32_t* key_cache_ptr = (uint32_t*) (key_cache + cache_offset);
    uint32_t* value_cache_ptr = (uint32_t*) (value_cache + cache_offset);

    #pragma unroll
    for (int k = 0; k < BLOCKSIZE_Z; k++)
    {
        *key_cache_ptr = *key_ptr;
        key_ptr += state_stride[0] / BLOCKSIZE_X;
        key_cache_ptr += cache_stride[0] / BLOCKSIZE_X;
    }
    #pragma unroll
    for (int k = 0; k < BLOCKSIZE_Z; k++)
    {
        *value_cache_ptr = *value_ptr;
        value_ptr += state_stride[0] / BLOCKSIZE_X;
        value_cache_ptr += cache_stride[0] / BLOCKSIZE_X;
    }
}

void q4_attn_cuda
(
    ExLlamaTuning* tuningParams,
    hipStream_t stream,
    hipblasHandle_t handle,
    half* x,
    const half* rms_norm_weight,    // shape == (x.shape[1],) == (dim,)
    float epsilon,
    half* query_states,
    half* key_states,
    half* value_states,
    Q4Matrix* q_proj,
    Q4Matrix* k_proj,
    Q4Matrix* v_proj,
    half* sin,
    half* cos,
    const int bsz,
    const int q_len,
    const int dim,
    const int head_dim,
    const int num_heads,
    const int num_kv_heads,
    const int past_len,
    half* key_cache,
    half* value_cache,
    const half* q_a,
    const half* q_b,
    const int q_rank,
    const half* k_a,
    const half* k_b,
    const int k_rank,
    const half* v_a,
    const half* v_b,
    const int v_rank,
    half* lora_temp,
    const int max_seq_len,
    const int device_index
)
{
    // Cache update grid

    dim3 threads(THREADS_X, THREADS_Y, THREADS_Z);

    dim3 blocks
    (
        ((head_dim + THREADS_X - 1) / THREADS_X + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
        q_len,
        ((num_kv_heads + THREADS_Z - 1) / THREADS_Z + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z
    );

    int _rows_per_batch = q_len * num_heads;
    int _rows_per_batch_kv = q_len * num_kv_heads;

    CudaBuffers* buffers = get_buffers(device_index);

    // Layernorm

    half* temp_x = buffers->temp_state + q_len * dim;
    rms_norm_cuda(tuningParams, x, rms_norm_weight, temp_x, epsilon, q_len, dim, device_index);

    // Adapters

    if (q_a)
    {
        half_matmul_cublas_cuda(tuningParams, temp_x, q_a, lora_temp, q_len, dim, q_rank, handle);
        half_matmul_cublas_cuda(tuningParams, lora_temp, q_b, query_states, q_len, q_rank, dim, handle);
    }
    if (k_a)
    {
        half_matmul_cublas_cuda(tuningParams, temp_x, k_a, lora_temp, q_len, dim, k_rank, handle);
        half_matmul_cublas_cuda(tuningParams, lora_temp, k_b, key_states, q_len, k_rank, dim, handle);
    }
    if (v_a)
    {
        half_matmul_cublas_cuda(tuningParams, temp_x, v_a, lora_temp, q_len, dim, v_rank, handle);
        half_matmul_cublas_cuda(tuningParams, lora_temp, v_b, value_states, q_len, v_rank, dim, handle);
    }

    if (!tuningParams->concurrent_streams)
    {
        // Project q, k, v

        q4_matmul_cuda(tuningParams, temp_x, q_len, q_proj, query_states, q_a ? true : false);
        q4_matmul_cuda(tuningParams, temp_x, q_len, k_proj, key_states, k_a ? true : false);
        q4_matmul_cuda(tuningParams, temp_x, q_len, v_proj, value_states, v_a ? true : false);

        // Positional embeddings q, k

        rope_cuda(tuningParams, query_states, sin, cos, bsz, _rows_per_batch, head_dim, num_heads, past_len);
        rope_cuda(tuningParams, key_states, sin, cos, bsz, _rows_per_batch_kv, head_dim, num_kv_heads, past_len);

        // Update cache tensors with projected k, v

        update_cache_kernel<<<blocks, threads>>>(key_states, value_states, key_cache, value_cache, head_dim, num_kv_heads, q_len, max_seq_len, past_len);
    }
    else
    {
        // Project q, k, v, add positional embeddings to q, k, update cache tensors with projected k, v

        hipStream_t str_1 = buffers->alt_stream_1;
        hipStream_t str_2 = buffers->alt_stream_2;
        hipStream_t str_3 = buffers->alt_stream_3;
        hipEvent_t sync_1 = buffers->alt_stream_1_done;
        hipEvent_t sync_2 = buffers->alt_stream_2_done;
        hipEvent_t sync_3 = buffers->alt_stream_3_done;

        // str_1: project q, positions q, sync

        q4_matmul_cuda(tuningParams, temp_x, q_len, q_proj, query_states, q_a ? true : false, str_1);
        rope_cuda(tuningParams, query_states, sin, cos,  bsz, _rows_per_batch, head_dim, num_kv_heads, past_len, str_1);
        hipEventRecord(sync_1, str_1);

        // str_2: project k, positions k, sync

        q4_matmul_cuda(tuningParams, temp_x, q_len, k_proj, key_states, k_a ? true : false, str_2);
        rope_cuda(tuningParams, key_states, sin, cos,  bsz, _rows_per_batch_kv, head_dim, num_kv_heads, past_len, str_2);
        hipEventRecord(sync_2, str_2);

        // str_3: project v, wait for str_2, copy (k,v) to cache, sync

        q4_matmul_cuda(tuningParams, temp_x, q_len, v_proj, value_states, v_a ? true : false, buffers->alt_stream_3);
        hipStreamWaitEvent(str_3, sync_2, 0);
        update_cache_kernel<<<blocks, threads, 0, str_3>>>(key_states, value_states, key_cache, value_cache, head_dim, num_kv_heads, q_len, max_seq_len, past_len);
        hipEventRecord(sync_3, str_3);

        // default: wait for str_1 and str_3

        hipStreamWaitEvent(NULL, sync_1, 0);
        hipStreamWaitEvent(NULL, sync_3, 0);
    }
}

void q4_attn_2_cuda
(
    ExLlamaTuning* tuningParams,
    hipblasHandle_t handle,
    half* x,
    half* attn_output,
    Q4Matrix* o_proj,
    const int height,
    const half* o_a,
    const half* o_b,
    const int o_rank,
    half* lora_temp
)
{
    if (o_a)
    {
        int dim = o_proj->height;
        half_matmul_cublas_cuda(tuningParams, attn_output, o_a, lora_temp, height, dim, o_rank, handle);
        half_matmul_cublas_cuda(tuningParams, lora_temp, o_b, x, height, o_rank, dim, handle, true);
    }

    q4_matmul_cuda(tuningParams, attn_output, height, o_proj, x, true);
}

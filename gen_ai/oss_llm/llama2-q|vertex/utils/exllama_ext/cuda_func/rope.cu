#include "hip/hip_runtime.h"
#include "rope.cuh"
#include "../util.cuh"
#include "../matrix.cuh"

const int THREADS_X = 32;
const int THREADS_Y = 4;
const int MAX_POS_EMBEDDINGS = 32768;  // Actual number doesn't matter

typedef void (*fp_rope_cuda_kernel)
(
    half*,
    const half*,
    const half*,
    int,
    int,
    int,
    int
);

template<bool use_half2>
__global__ void rope_cuda_kernel
(
    half* __restrict__ x,
    const half* __restrict__ sin,
    const half* __restrict__ cos,
    int rows_per_batch,
    int head_dim,
    int num_heads,
    int past_len
)
{
    // These heights aren't used so it's okay if they're wrong.
    MatrixView_half_rw x_(x, rows_per_batch, head_dim);
    MatrixView_half sin_(sin, MAX_POS_EMBEDDINGS, head_dim);
    MatrixView_half cos_(cos, MAX_POS_EMBEDDINGS, head_dim);

    int column = (blockIdx.x * THREADS_X + threadIdx.x); if constexpr (use_half2) column *= 2;
    int half_dim = head_dim / 2;
    if (column >= half_dim) return;

    int row = blockIdx.y * THREADS_Y + threadIdx.y;
    if (row >= rows_per_batch) return;
    int batch_offset = blockIdx.z * rows_per_batch;
    int row_offset = batch_offset + row;

    // Get sin and cos

    int sincos_row = past_len + row / num_heads;

    if constexpr (use_half2)
    {
        half2 cos2_l = cos_.item_half2(sincos_row, column);
        half2 cos2_r = cos_.item_half2(sincos_row, column + half_dim);
        half2 sin2_l = sin_.item_half2(sincos_row, column);
        half2 sin2_r = sin_.item_half2(sincos_row, column + half_dim);
        sin2_l = __hneg2(sin2_l);

        // Apply embedding to row

        half2 item2_l = x_.item_half2(row_offset, column);
        half2 item2_r = x_.item_half2(row_offset, column + half_dim);
        half2 item2_ls = __hmul2(item2_r, sin2_l);
        half2 item2_rs = __hmul2(item2_l, sin2_r);
        item2_l = __hfma2(item2_l, cos2_l, item2_ls);
        item2_r = __hfma2(item2_r, cos2_r, item2_rs);
        x_.set_half2(row_offset, column, item2_l);
        x_.set_half2(row_offset, column + half_dim, item2_r);
    }
    else
    {
        half cos_l = cos_.item(sincos_row, column);
        half cos_r = cos_.item(sincos_row, column + half_dim);
        half sin_l = sin_.item(sincos_row, column);
        half sin_r = sin_.item(sincos_row, column + half_dim);
        sin_l = __hneg(sin_l);

        // Apply embedding to row

        half item_l = x_.item(row_offset, column);
        half item_r = x_.item(row_offset, column + half_dim);
        half item_ls = __hmul(item_r, sin_l);
        half item_rs = __hmul(item_l, sin_r);
        item_l = __hfma(item_l, cos_l, item_ls);
        item_r = __hfma(item_r, cos_r, item_rs);
        x_.set(row_offset, column, item_l);
        x_.set(row_offset, column + half_dim, item_r);
    }
}

fp_rope_cuda_kernel rope_cuda_kernel_pick(ExLlamaTuning* tuningParams)
{
    // <bool use_half2>
    if (tuningParams->matmul_no_half2) {
        return rope_cuda_kernel<false>;
    } else {
        return rope_cuda_kernel<true>;
    }
};

void rope_cuda
(
    ExLlamaTuning* tuningParams,
    half* x,
    const half* sin,
    const half* cos,
    const int bsz,
    const int rows_per_batch,
    const int head_dim,
    const int num_heads,
    const int past_len,
    hipStream_t alt_stream
)
{
    dim3 threads(THREADS_X, THREADS_Y, 1);

    dim3 blocks
    (
        (head_dim + THREADS_X - 1) / THREADS_X / 2 / (tuningParams->rope_no_half2 ? 1 : 2),
        (rows_per_batch + THREADS_Y - 1) / THREADS_Y,
        int(bsz)
    );

    fp_rope_cuda_kernel kernel = rope_cuda_kernel_pick(tuningParams);
    kernel<<<blocks, threads, 0, alt_stream>>>(x, sin, cos, rows_per_batch, head_dim, num_heads, past_len);
}
